/*
 * Copyright (c) 2010-2011
 *      RIKEN
 * 	All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 */
/*
  Contributed by Takao, Yasuyoshi and Nakata, Maho, 2010-2011
*/

void Rgemm_gpu(const char *transa, const char *transb, mpackint m, mpackint n, mpackint k, dd_real alpha, dd_real * Adev, mpackint lda, dd_real * Bdev, mpackint ldb, dd_real beta, dd_real * Cdev, mpackint ldc)
{
    mpackint nota, notb;
    hipError_t rc;

    nota = Mlsame_dd(transa, "N");
    notb = Mlsame_dd(transb, "N");

    if (notb) {
        if (nota) {
	    //Form C := alpha*A*B + beta*C.
            // calculating and updating C
            dim3 grid(m / Bm + (m % Bm == 0 ? 0 : 1), n / (Gn * Bn)
                      + (n % (Gn * Bn) == 0 ? 0 : 1)), block(Bm, Bn);
            if(m % Bm == 0 && k % Bk == 0 && n % (Gn * Bn) == 0){
                CalcC_NN0 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }else{
                CalcC_NN1 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }
        } else {
	    //Form C := alpha*A'*B + beta*C.
            // calculating and updating C
            dim3 grid(m / Bm + (m % Bm == 0 ? 0 : 1), n / (Gn * Bn)
                      + (n % (Gn * Bn) == 0 ? 0 : 1)), block(Bm, Bn);
            if(m % Bm == 0 && k % Bk == 0 && n % (Gn * Bn) == 0){
                CalcC_TN0 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }else{
                CalcC_TN1 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }
        }
    } else {
        if (nota) {
	    //Form  C := alpha*A*B' + beta*C.
            // calculating and updating C
            dim3 grid(m / Bm + (m % Bm == 0 ? 0 : 1), n / (Gn * Bn)
                      + (n % (Gn * Bn) == 0 ? 0 : 1)), block(Bm, Bn);
            if(m % Bm == 0 && k % Bk == 0 && n % (Gn * Bn) == 0){
                CalcC_NT0 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }else{
                CalcC_NT1 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }
        } else {
	    //Form  C := alpha*A'*B' + beta*C.
            // calculating and updating C
            dim3 grid(m / Bm + (m % Bm == 0 ? 0 : 1), n / (Gn * Bn)
                      + (n % (Gn * Bn) == 0 ? 0 : 1)), block(Bm, Bn);
            if(m % Bm == 0 && k % Bk == 0 && n % (Gn * Bn) == 0){
                CalcC_TT0 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }else{
                CalcC_TT1 <<< grid, block >>> (Adev, Bdev, Cdev, m, n, k, lda, ldb, ldc, alpha, beta);
            }
        }
    }
//    hipDeviceSynchronize();
}

