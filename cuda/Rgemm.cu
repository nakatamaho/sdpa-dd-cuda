#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2010-2011
 *      RIKEN
 * 	All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 */
/*
  Contributed by Takao, Yasuyoshi and Nakata, Maho, 2010-2011
*/
/*
Based on http://www.netlib.org/blas/dgemm.f
Rgemm performs one of the matrix-matrix operations
 C := alpha*op(A)*op(B) + beta*C,
where op(X) is one of
 op(X) = X or op(X) = X',
alpha and beta are scalars, and A, B and C are matrices, with op( A )
an m by k matrix, op(B) a k by n matrix and C an m by n matrix.
*/

#include <iostream>
#include <iomanip>
#include <stdio.h>
#include "mpack/mpack_config.h"

int Mlsame_dd(const char *a, const char *b);
void Mxerbla_dd(const char *srname, int info);

#define __RGEMM_INSIDE_KERNEL__
#include "dd_real_cuda.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// matrix block size
#define Bm  (16)
#define Bk  (16)
#define Bn  (16)
#define Gn   (4)

#define fetch_x_A(i) Adev[(i)]
#define fetch_x_B(i) Bdev[(i)]

//for alpha*A*B + beta
__global__ void CalcC_NN0 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);
__global__ void CalcC_NN1 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);

//for alpha*A'*B + beta
__global__ void CalcC_TN0 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);
__global__ void CalcC_TN1 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);

//for alpha*A*B' + beta
__global__ void CalcC_NT0 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);
__global__ void CalcC_NT1 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);

//for alpha*A'*B' + beta
__global__ void CalcC_TT0 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);
__global__ void CalcC_TT1 (dd_real * Adev, dd_real * Bdev, dd_real * Cdev, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc, dd_real alpha, dd_real beta);

void Is_cuda_Rgemm_error(hipError_t rc, const char *mes, mpackint m, mpackint n, mpackint k, mpackint lda, mpackint ldb, mpackint ldc)
{
    if (rc != hipSuccess) {
	fprintf(stderr, "%s : m:%d n:%d k:%d lda:%d ldb:%d ldc:%d\n", mes, m, n, k, lda, ldb, ldc);
	exit(1);
    }
    /* not an error */
}

#include "Rgemm_gpu.cu"
#include "Rgemm_NN_0.cu"
#include "Rgemm_NN_p.cu"
#include "Rgemm_TN_0.cu"
#include "Rgemm_TN_p.cu"
#include "Rgemm_NT_0.cu"
#include "Rgemm_NT_p.cu"
#include "Rgemm_TT_0.cu"
#include "Rgemm_TT_p.cu"

static double ops_counter = 0.0;

//allocate device memory for GPU
static dd_real *Adev, *Bdev, *Cdev = NULL;
static int size_A_g, size_B_g, size_C_g;

void Alloc(int size_A, int size_B, int size_C, int m, int n, int k, int lda, int ldb, int ldc)
{
  hipError_t rc;
  // first alloc
  if (Cdev == NULL) {
    size_A_g = size_A*2;
    size_B_g = size_B*2;
    size_C_g = size_C*2;
    rc = hipMalloc((void **) &Adev, size_A_g * sizeof(dd_real));
    Is_cuda_Rgemm_error(rc, "hipMalloc A error", m, n, k, lda, ldb, ldc);
    rc = hipMalloc((void **) &Bdev, size_B_g * sizeof(dd_real));
    Is_cuda_Rgemm_error(rc, "hipMalloc B error", m, n, k, lda, ldb, ldc);
    rc = hipMalloc((void **) &Cdev, size_C_g * sizeof(dd_real));
    Is_cuda_Rgemm_error(rc, "hipMalloc C error", m, n, k, lda, ldb, ldc);
  }

  if (size_A_g < size_A) {
    rc = hipFree(Adev);
    size_A_g = size_A*2;
    rc = hipMalloc((void **) &Adev, size_A_g * sizeof(dd_real));
    Is_cuda_Rgemm_error(rc, "hipMalloc A error", m, n, k, lda, ldb, ldc);
  }

  if (size_B_g < size_B) {
    rc = hipFree(Bdev);
    size_B_g = size_B*2;
    rc = hipMalloc((void **) &Bdev, size_B_g * sizeof(dd_real));
    Is_cuda_Rgemm_error(rc, "hipMalloc B error", m, n, k, lda, ldb, ldc);
  }

  if (size_C_g < size_C) {
    rc = hipFree(Cdev);
    size_C_g = size_C*2;
    rc = hipMalloc((void **) &Cdev, size_C_g * sizeof(dd_real));
    Is_cuda_Rgemm_error(rc, "hipMalloc C error", m, n, k, lda, ldb, ldc);
  }
}



void Rgemm_X(const char *transa, const char *transb, mpackint m, mpackint n, mpackint k, dd_real alpha, dd_real * A, mpackint lda, dd_real * B, mpackint ldb, dd_real beta, dd_real * C, mpackint ldc)
{
    mpackint i, j, nota, notb, nrowa, nrowb, ncola, info;
    dd_real temp, Zero, One;
    hipError_t rc;

    dd_set(Zero, 0.0, 0.0);
    dd_set(One, 1.0, 0.0);

    nota = Mlsame_dd(transa, "N");
    notb = Mlsame_dd(transb, "N");
    if (nota) {
	nrowa = m;
	ncola = k;
    } else {
	nrowa = k;
	ncola = m;
    }
    if (notb) {
	nrowb = k;
    } else {
	nrowb = n;
    }
//Test the input parameters.
    info = 0;
    if (!nota && (!Mlsame_dd(transa, "C"))
	&& (!Mlsame_dd(transa, "T")))
	info = 1;
    else if (!notb && (!Mlsame_dd(transb, "C"))
	     && (!Mlsame_dd(transb, "T")))
	info = 2;
    else if (m < 0)
	info = 3;
    else if (n < 0)
	info = 4;
    else if (k < 0)
	info = 5;
    else if (lda < std::max((mpackint) 1, nrowa))
	info = 8;
    else if (ldb < std::max((mpackint) 1, nrowb))
	info = 10;
    else if (ldc < std::max((mpackint) 1, m))
	info = 13;
    if (info != 0) {
	Mxerbla_dd("Rgemm ", info);
	return;
    }
    //Quick return if possible.
    if ((m == 0)
	|| (n == 0)
	|| ((dd_eq(alpha, Zero)
	     || (k == 0))
	    && dd_eq(beta, One))) {
	return;
    }

    //And when alpha == 0.0
    if (dd_eq(alpha, Zero)) {
	if (dd_eq(beta, Zero)) {
	    for (j = 0; j < n; j++) {
		for (i = 0; i < m; i++) {
		    C[i + j * ldc] = Zero;
		}
	    }
	} else {
	    for (j = 0; j < n; j++) {
		for (i = 0; i < m; i++) {
		    dd_mul_host(beta, C[i + j * ldc], C[i + j * ldc]);
		}
	    }
	}
	return;
    }

    int size_A, size_B, size_C;
    if (nota)
	size_A = lda * k - (lda - m);
    else
	size_A = lda * m - (lda - k);
    if (notb)
	size_B = ldb * n - (ldb - k);
    else
	size_B = ldb * k - (ldb - n);
    size_C = ldc * n - (ldc - m);

    Alloc(size_A, size_B, size_C, m, n, k, lda, ldb, ldc);
    
    rc = hipMemcpy(Adev, A, size_A * sizeof(dd_real), hipMemcpyHostToDevice);
        Is_cuda_Rgemm_error(rc, "hipMemcpy A error", m, n, k, lda, ldb, ldc);
    rc = hipMemcpy(Bdev, B, size_B * sizeof(dd_real), hipMemcpyHostToDevice);
        Is_cuda_Rgemm_error(rc, "hipMemcpy B error", m, n, k, lda, ldb, ldc);
    rc = hipMemcpy(Cdev, C, size_C * sizeof(dd_real), hipMemcpyHostToDevice);
        Is_cuda_Rgemm_error(rc, "hipMemcpy C error", m, n, k, lda, ldb, ldc);

    Rgemm_gpu(transa, transb, m, n, k, alpha, Adev, lda, Bdev, ldb, beta, Cdev, ldc);

    rc = hipMemcpy(C, Cdev, size_C * sizeof(dd_real), hipMemcpyDeviceToHost);
        Is_cuda_Rgemm_error(rc, "hipMemcpy C error", m, n, k, lda, ldb, ldc);

    return;
}
